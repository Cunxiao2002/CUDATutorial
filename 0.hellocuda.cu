#include<stdio.h>
#include<hip/hip_runtime.h>


__global__ void hello_cuda(){
    unsigned int idx = blockDim.x * blockIdx.x + threadIdx.x;
    printf("block id = [%d], thread id = [%d], hello cuda\n", blockIdx.x, idx);
}

int main(){
    hello_cuda<<<1,1>>>();
    hipDeviceSynchronize();
    return 0;
}